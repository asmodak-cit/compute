/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < m) && (col < n)) {

        float sum = 0;
        for (int i = 0; i < k; i++) {            
            sum += A[row*k + i] * B[i*n + col];
        }

        C[row * n + col] = sum;

    }
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 16; // Use 16x16 thread blocks

    //INSERT CODE HERE

    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);  // Each block will have BLOCK_SIZE threads in each dimension
    dim3 dim_grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm<<<dim_grid, dim_block>>>(m, n, k, A, B, C);
}


